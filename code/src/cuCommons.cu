#include "commons.h"
#include "cuCommons.h"

void CudaAssert(hipError_t error, const char *code, const char *file, int line) {
  if (error != hipSuccess) {
    cerr << "Cuda error :" << code << ", " << file << ":" << error << endl;
    exit(1);
  }
}

void printArray(int *dev, int n, context_t &context) {
  transform(
      [=] MGPU_DEVICE(int i) {
        for (int a = 0; a < n; a++) {
          printf("%d", dev[a]);
        }
        printf("\n");
      },
      1, context);
  context.synchronize();
}

CuGraph::CuGraph(const Graph &G, context_t &context) : n(G.n), context(context) {
  CUCHECK(hipMalloc((void **)&devMatrix, sizeof(int) * n * n));
  CUCHECK(hipMalloc((void **)&devFirstNeighbor, sizeof(int) * n));
  CUCHECK(hipMalloc((void **)&devNextNeighbor, sizeof(int) * n * n));

  CUCHECK(hipMemcpy(devFirstNeighbor, G._first_neighbour.data(), sizeof(int) * n, hipMemcpyHostToDevice));

  for (int i = 0; i < n; i++) {
    CUCHECK(hipMemcpy(devMatrix + (i * n), G._matrix[i].data(), sizeof(int) * n, hipMemcpyHostToDevice));
    CUCHECK(hipMemcpy(devNextNeighbor + (i * n), G._next_neighbour[i].data(), sizeof(int) * n,
                       hipMemcpyHostToDevice));
  }
}